#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <glm/glm.hpp>

#include "jfa.h"
#include "misc.h"

using namespace glm;

constexpr const int invalid_pointer = 1<<30;

__device__
int coord_to_id(ivec2 coord, int w) {
    return coord.x + coord.y*w;
}

__device__
ivec2 id_to_coord(int id, int w) {
    return ivec2(id % w, id / w);
}

__device__
bool is_coord_in_bounds(ivec2 coord, int w) {
    return 0 <= coord.x && 0 <= coord.y && coord.x < w && coord.y < w;
}

__device__
bool is_id_in_bounds(int id, int w) {
    return 0 <= id && id < w*w;
}

__device__
ivec2 wrap_coord_clamp(ivec2 coord, int w) {
    return clamp(coord, ivec2(0), ivec2(w-1));
}

__device__
ivec2 wrap_coord_repeat(ivec2 coord, int w) {
    return (coord + ivec2(w)) % ivec2(w);
}

__device__
__forceinline__
int transpose_id(int id, int w) {
    if (id == invalid_pointer)
        return invalid_pointer;

    ivec2 coord = id_to_coord(id, w);
    return coord_to_id(ivec2(coord.y, coord.x), w);
}

__global__
void transpose_pointers(int *dst, int *src, int w) {
    unsigned int o0 = blockIdx.x * 32 + blockIdx.y * 32 * w;
    unsigned int o1 = blockIdx.y * 32 + blockIdx.x * 32 * w;

    __shared__ int data[32*32];

    for (int y = 0; y < 32; y += blockDim.y) {
        int pi = threadIdx.x + (y + threadIdx.y) * w;
        int di = threadIdx.y + y + threadIdx.x * 32;

        int p = src[pi + o0];
        data[di] = transpose_id(p, w);
    }

    __syncthreads();

    for (int y = 0; y < 32; y += blockDim.y) {
        int pi = threadIdx.x + (y + threadIdx.y) * w;
        int di = threadIdx.x + (y + threadIdx.y) * 32;

        dst[pi + o1] = data[di];
    }
}

__global__
extern void jfa_init_pointers(int *pointers, bool *input, int w) {
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    pointers[gid] = input[gid] ? gid : invalid_pointer;
}

__global__
void jfa_to_sdf(float *sdf, int *pointers, int w) {
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    auto a = vec2(id_to_coord(gid, w));
    auto b = vec2(id_to_coord(pointers[gid], w));
    // sdf[gid] = length(b - a);
    sdf[gid] = length(b - a) / (w * 2.0f * sqrt(2.0f));
}

__global__
void jfa_impl_0(int *pointers, int w, int s) {
    int gid = blockIdx.x*blockDim.x + threadIdx.x;

    ivec2 tcoord = id_to_coord(gid, w);
    int tpid = pointers[gid]; // loading early hides latency
    float tl = w*w*2;

    for (int y = -1; y < 2; ++y) {
        for (int x = -1; x < 2; ++x) {
            // if (x == 0 && y == 0)
            //     continue;

            // ivec2 coord = wrap_coord_repeat(tcoord + ivec2(x, y) * s, w);
            ivec2 coord = tcoord + ivec2(x, y) * s;
            if (!is_coord_in_bounds(coord, w))
                continue;

            // ivec2 coord = id_to_coord((gid + y*w*s + x*s) % (w*w), w);
            int id = coord_to_id(coord, w);
            int pid = pointers[id];

            if (pid != invalid_pointer) {
                ivec2 pcoord = id_to_coord(pid, w);
                auto v = pcoord - tcoord;
                float l = (v.x*v.x + v.y*v.y);

                if (tpid == invalid_pointer || l < tl) {
                    tpid = pid;
                    tl = l;
                }
            }
        }
    }

    pointers[gid] = tpid;
}

void jfa_0(unsigned int B, unsigned int T, int *pointers, int w) {
    int s = w/2;
    while (s > 0) {
        jfa_impl_0<<<B, T>>>(pointers, w, s);
        s /= 2;
    }
}

template <int X, int Y>
__global__
void jfa_impl_3(int *pointers, int w, int s) {
    int gid = blockIdx.x*blockDim.x + threadIdx.x;

    ivec2 tcoord = id_to_coord(gid, w);
    int tpid = pointers[gid]; // loading early hides latency
    float tl = w*w*2;

    for (int i = -1; i < 2; ++i) {
        ivec2 coord = tcoord + ivec2(i * X, i * Y) * s;
        if (!is_coord_in_bounds(coord, w))
            continue;

        int id = coord_to_id(coord, w);
        int pid = pointers[id];

        if (pid != invalid_pointer) {
            ivec2 pcoord = id_to_coord(pid, w);
            auto v = pcoord - tcoord;
            float l = (v.x*v.x + v.y*v.y);

            if (tpid == invalid_pointer || l < tl) {
                tpid = pid;
                tl = l;
            }
        }
    }

    pointers[gid] = tpid;
}

// __device__
// void jfa_impl_4_horizontal_warp(volatile int *pids, int w, int s) {
//     int gid = blockIdx.x*blockDim.x + threadIdx.x;
//     int tid = threadIdx.x;
// }

__device__
__forceinline__
void jfa_impl_4_horizontal_check(int *pids, ivec2 tcoord, int &tpid, float &tl, int w, int s, int blockSize) {
    int tid = threadIdx.x;

    int pid = pids[tid + blockSize + s];

    ivec2 pcoord = id_to_coord(pid, w);
    auto v = pcoord - tcoord;
    float l = length_squared(v);

    if (l < tl) {
        tpid = pid;
        tl = l;
    }
}

__device__
__forceinline__
void jfa_impl_4_horizontal_do(int *pids, ivec2 tcoord, int &tpid, float &tl, int w, int s, int blockSize) {
    int tid = threadIdx.x;
    jfa_impl_4_horizontal_check(pids, tcoord, tpid, tl, w, -s, blockSize);
    jfa_impl_4_horizontal_check(pids, tcoord, tpid, tl, w,  s, blockSize);
    pids[tid + blockSize] = tpid;
    __syncthreads();
}

template <unsigned int blockSize>
__global__
void jfa_impl_4_horizontal(int *pointers, int w) {
    int gid = blockIdx.x*blockSize + threadIdx.x;
    int tid = threadIdx.x;

    __shared__ int pids[blockSize * 3];

    pids[tid]       = pointers[(gid - blockSize + w*w) % (w*w)];
    pids[tid + blockSize]   = pointers[gid];
    pids[tid + blockSize*2] = pointers[(gid + blockSize) % (w*w)];

    __syncthreads();

    ivec2 tcoord = id_to_coord(gid, w);
    int tpid = pids[tid + blockSize];
    float tl = w*w*2.0f;
    if (tpid != invalid_pointer) {
        ivec2 pcoord = id_to_coord(tpid, w);
        auto v = pcoord - tcoord;
        float l = length_squared(v);
        tl = l;
    }

    if constexpr (blockSize >= 128) {
        jfa_impl_4_horizontal_do(pids, tcoord, tpid, tl, w, 128, blockSize);
    }
    if constexpr (blockSize >= 64) {
        jfa_impl_4_horizontal_do(pids, tcoord, tpid, tl, w, 64, blockSize);
    }
    if constexpr (blockSize >= 32) {
        jfa_impl_4_horizontal_do(pids, tcoord, tpid, tl, w, 32, blockSize);
    }
    if constexpr (blockSize >= 16) {
        jfa_impl_4_horizontal_do(pids, tcoord, tpid, tl, w, 16, blockSize);
    }
    if constexpr (blockSize >= 8) {
        jfa_impl_4_horizontal_do(pids, tcoord, tpid, tl, w, 8, blockSize);
    }
    if constexpr (blockSize >= 4) {
        jfa_impl_4_horizontal_do(pids, tcoord, tpid, tl, w, 4, blockSize);
    }
    if constexpr (blockSize >= 2) {
        jfa_impl_4_horizontal_do(pids, tcoord, tpid, tl, w, 2, blockSize);
    }
    if constexpr (blockSize >= 1) {
        jfa_impl_4_horizontal_do(pids, tcoord, tpid, tl, w, 1, blockSize);
    }

    pointers[gid] = tpid;
}

void jfa_2(unsigned int B, unsigned int T, int *pointers, int w) {
    int s = w/2;
    while (0 < s) {
        jfa_impl_3<1, 0><<<B, T>>>(pointers, w, s);
        s /= 2;
    }
    s = w/2;
    while (0 < s) {
        jfa_impl_3<0, 1><<<B, T>>>(pointers, w, s);
        s /= 2;
    }
}

void jfa_3(unsigned int B, unsigned int T, int *pointers, int w) {
    assert(w % T == 0);
    assert(T == 128);

    int s = w/2;
    while (T < s) {
        std::cout << "jfa_impl_3<1, 0> s=" << s << " ms=" << perf([&] {
            jfa_impl_3<1, 0><<<B, T>>>(pointers, w, s);
        }) << std::endl;
        s /= 2;
    }
    std::cout << "jfa_impl_4_horizontal s=" << s << " ms=" << perf([&] {
        jfa_impl_4_horizontal<128><<<B, T>>>(pointers, w);
    }) << std::endl;

    float ps = 0.0f;
    s = w/2;
    while (0 < s) {
        float p = perf([&] {
            jfa_impl_3<0, 1><<<B, T>>>(pointers, w, s);
        });
        std::cout << "jfa_impl_3<0, 1> s=" << s << " ms=" << p << std::endl;
        ps += p;
        s /= 2;
    }
    std::cout << "ps: " << ps << std::endl;
}

void jfa_4(unsigned int B, unsigned int T, int *pointers, int w) {
    assert(w % T == 0);
    assert(T == 128);

    float perfsum = 0.0f;

    int s = w/2;
    while (T <= s) {
        auto p = perf([&] {
            jfa_impl_3<1, 0><<<B, T>>>(pointers, w, s);
        });
        perfsum += p;
        std::cout << "jfa_impl_3<1, 0> s=" << s << " ms=" << p << std::endl;
        s /= 2;
    }
    auto p = perf([&] {
        jfa_impl_4_horizontal<128><<<B, T>>>(pointers, w);
    });
    perfsum += p;
    std::cout << "jfa_impl_4_horizontal s=" << s << " ms=" << p << std::endl;

    int *pt;
    hipMalloc(&pt, w*w * sizeof(int));
    p = perf([&] {
        auto dimBlock = dim3(32, 4);
        auto dimGrid = dim3(w / 32, w / 32);
        transpose_pointers<<<dimGrid, dimBlock>>>(pt, pointers, w);
    });
    std::swap(pt, pointers);
    perfsum += p;
    std::cout << "transpose_pointers ms=" << p << std::endl;

    s = w/2;
    while (T <= s) {
        auto p = perf([&] {
            jfa_impl_3<1, 0><<<B, T>>>(pointers, w, s);
        });
        perfsum += p;
        std::cout << "jfa_impl_3<1, 0> s=" << s << " ms=" << p << std::endl;
        s /= 2;
    }
    p = perf([&] {
        jfa_impl_4_horizontal<128><<<B, T>>>(pointers, w);
    });
    perfsum += p;
    std::cout << "jfa_impl_4_horizontal s=" << s << " ms=" << p << std::endl;

    std::cout << perfsum << std::endl;

    p = perf([&] {
        auto dimBlock = dim3(32, 4);
        auto dimGrid = dim3(w / 32, w / 32);
        transpose_pointers<<<dimGrid, dimBlock>>>(pt, pointers, w);
    });
    std::swap(pt, pointers);
    hipFree(pt);
    perfsum += p;
    std::cout << "transpose_pointers ms=" << p << std::endl;

    CHECK_LAST_CUDA_ERROR();
}
