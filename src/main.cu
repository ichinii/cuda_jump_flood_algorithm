#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <memory>
#include <chrono>

#include "jfa.h"
#include "display.h"
#include "misc.h"

// constexpr const unsigned int W = 512;
constexpr const unsigned int W = 1<<10;
constexpr const unsigned int S = W*W;

constexpr const unsigned int T = std::min(128u, W);
constexpr const unsigned int B = S/T;
static_assert(T * B == S);

// A single iteration of Bob Jenkins' One-At-A-Time hashing algorithm.
__device__
unsigned int hash(unsigned int x) {
    x += ( x << 10u );
    x ^= ( x >>  6u );
    x += ( x <<  3u );
    x ^= ( x >> 11u );
    x += ( x << 15u );
    return x;
}

__global__
void create_input(bool *input, float seed) {
    unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;
    auto y = gid / W;
    input[gid] = hash(gid + y*0.1f) % 50000 == 0;
}

__global__
void visualize(
    glm::vec4 *output,
    [[maybe_unused]] bool *input,
    [[maybe_unused]] int *pointers,
    [[maybe_unused]] float *sdf,
    glm::ivec2 viewport
) {
    unsigned int gid = blockIdx.x*blockDim.x + threadIdx.x;
    auto uv = glm::ivec2(gid % viewport.x, gid / viewport.x) % glm::ivec2(W);
    auto i = uv.x + uv.y * W;
    [[maybe_unused]] auto l = sdf[i];
    [[maybe_unused]] auto b = input[i];
    [[maybe_unused]] auto p = pointers[i] / static_cast<float>(S);

    if (gid < viewport.x*viewport.y) {
        float h = (hash(pointers[i]      ) % S) / static_cast<float>(S);
        float s = (hash(pointers[i] + S  ) % S) / static_cast<float>(S) * 0.8f + 0.2f;
        float v = (hash(pointers[i] + S*2) % S) / static_cast<float>(S) * 0.8f + 0.2f;
        output[gid] = glm::vec4 (
            hsv2rgb(glm::vec3(h, !b && uv.x % 128 != 0 && uv.y % 128 != 0, 0.3f + b * 0.7f)),
            // 1.0f - l * 100.0f,
            // b,
            // p,
            1.f
        );
    }
}

int main()
{
    std::cout << "width: " << W << std::endl;
    std::cout << "pixels: " << S << std::endl;

    // device resources
    bool *input;
    int *pointers;
    int *pointers2;
    float *sdf;
    glm::vec4 *output;

    // host resources
    auto output_h = std::vector<glm::vec4>();

    auto resources_created = false;
    auto create_resources = [&] (glm::ivec2 viewport) {
        assert(!resources_created);

        hipMalloc(&input, S * sizeof(bool));
        hipMalloc(&pointers, S * sizeof(int));
        hipMalloc(&pointers2, S * sizeof(int));
        hipMalloc(&sdf, S * sizeof(float));
        hipMalloc(&output, viewport.x * viewport.y * sizeof(glm::vec4));
        output_h.resize(viewport.x * viewport.y);
        hipDeviceSynchronize();

        resources_created = true;
    };

    auto destroy_resources = [&] {
        if (resources_created) {
            hipDeviceSynchronize();
            hipFree(input);
            hipFree(pointers);
            hipFree(pointers2);
            hipFree(sdf);
            hipFree(output);
        }
        resources_created = false;
    };

    auto update_input = [&] ([[maybe_unused]] float elapsed_time) {
        create_input<<<B, T>>>(input, elapsed_time);
    };

    auto perf_elapsed = 0.0f;
    auto perf_count = 0.0f;
    auto update_sdf = [&] () {
        jfa_init_pointers<<<B, T>>>(pointers, input, W);
        // jfa_init_pointers_2D<<<B, T>>>((IdVec2*)pointers, input, W);
        perf_elapsed += perf( [&] {
            // jfa_2(B, T, pointers, W);
            jfa_6(B, T, pointers, pointers2, W);
            // jfa_5(B, T, (IdVec2*)pointers, W);
        } );
        ++perf_count;
        jfa_to_sdf<<<B, T>>>(sdf, pointers, W);
    };

    auto update_output = [&] (glm::ivec2 viewport) {
        // TODO: use cuda opengl interop to reduce copies
        auto s = viewport.x * viewport.y;
        int t = 128;
        int b = (s-1)/t+1;
        visualize<<<b, t>>>(output, input, pointers, sdf, viewport);
        hipMemcpy(output_h.data(), output, s * sizeof(glm::vec4), hipMemcpyDeviceToHost);
    };

    using namespace std::chrono;
    auto start_time = steady_clock::now();

    auto prev_viewport = glm::ivec2 {-1, -1};
    auto update = [&] (glm::ivec2 viewport) -> glm::vec4* {
        if (viewport != prev_viewport) {
            destroy_resources();
            create_resources(viewport);

            prev_viewport = viewport;
        }

            auto now = steady_clock::now();
            auto elapsed_time = duration_cast<milliseconds>(now - start_time).count() / 1000.f;

            update_input(elapsed_time);
            update_sdf();
            update_output(viewport);

            CHECK_LAST_CUDA_ERROR();

        return output_h.data();
    };
    display(update);

    destroy_resources();

    std::cout << (perf_elapsed / perf_count) << std::endl;

    return 0;
}
